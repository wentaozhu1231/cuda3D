#include "hip/hip_runtime.h"
/*
 * Authored by: Chen, Shifu
 * 
 * Email: chen@gmtk.org / sf.chen@ieee.org
 *
 * The code is distributed under BSD license, you are allowed to use, modify or sell this code, but a statement is required if you used this code any where.
 * 
 */
#include <stdio.h>
#include <stdlib.h>

#include <cutil.h>

#include "hip/hip_vector_types.h"

#include "bbsort.cuh"

#include "bbsort_kernel.cu"


float cmpKernel(float4 v){
	return v.w;
}



bool assignSliceToBuckets(unsigned int* sliceCount,int sliceSize,unsigned int* bucketOffset,unsigned int* bucketOfSlice,unsigned int* bucketSizes,unsigned int* sliceOffsetInBucket,int& bucketsCount,float step)
{
	int i=0;

	bool overflow=false;

	int tmpSum=0;

	bucketOffset[0]=0;

	for(i=0;i<sliceSize; i++){
		if(sliceCount[i] >BLOCK_SIZE)
		{
			overflow=true;
		}

		tmpSum += sliceCount[i];
		bucketOfSlice[i]=bucketsCount;
		bucketSizes[bucketsCount] = tmpSum;
		sliceOffsetInBucket[i]=tmpSum -sliceCount[i];
		if(tmpSum > BLOCK_SIZE )
		{	
			if(i != 0)
			{
				bucketOfSlice[i]=bucketsCount+1;
				bucketSizes[bucketsCount] -= sliceCount[i];
				sliceOffsetInBucket[i]=0;
				bucketOffset[bucketsCount+1]=bucketOffset[bucketsCount] + tmpSum -  sliceCount[i];

				bucketsCount++;
				tmpSum=sliceCount[i];
				bucketSizes[bucketsCount] = tmpSum;
			}
			else 
			{
				bucketOffset[bucketsCount+1]=bucketOffset[bucketsCount] + tmpSum ;
				sliceOffsetInBucket[i]=0;
				tmpSum=0;
				bucketsCount++;
			}
		}

	}
	bucketsCount++;

	return overflow;

}

void reduceMinMax(float4* dData,int size,float& result,bool isMax)
{

	int step;
	step=(size%2==0)?
		(size/2):(size/2 +1);

	int blockSize=BLOCK_SIZE;

	int blockCount;

	int length=size;

	float4 float4result;

	while(step > 0)
	{
		if(step%BLOCK_SIZE==0)
			blockCount=step/BLOCK_SIZE;
		else 
			blockCount=step/BLOCK_SIZE+1;

		if(isMax)
			reduceMaxD<<<blockCount,blockSize>>>(dData,step,length);
		else 
			reduceMinD<<<blockCount,blockSize>>>(dData,step,length);

		length=step;

		step=(step%2==0 || step==1)?(step/2):(step/2 +1);
	}

	CUDA_SAFE_CALL(hipMemcpy(&float4result, dData, sizeof(float4), hipMemcpyDeviceToHost));

	result=cmpKernel(float4result);
}

void evaluateDisorder(float4* dData,int size,float maxValue, float minValue, int& listOrder)
{
	int blockCount;

	if((size-1) % BLOCK_SIZE ==0)blockCount=size/BLOCK_SIZE;
	else blockCount=size/BLOCK_SIZE+1;

	float* dDiffData;
	CUDA_SAFE_CALL(hipMalloc((void**)&dDiffData, sizeof(float) * size));

	calDifferenceD<<<blockCount,BLOCK_SIZE,(BLOCK_SIZE)*sizeof(float4)>>>(dData,dDiffData,size);


	float sum=0;

	int step;
	step=(size%2==0)?
		(size/2):(size/2 +1);

	int blockSize=BLOCK_SIZE;

	int length=size;

	while(step > 0)
	{

		if(step%BLOCK_SIZE==0)
			blockCount=step/BLOCK_SIZE;
		else 
			blockCount=step/BLOCK_SIZE+1;

		reduceSumD<<<blockCount,blockSize>>>(dDiffData,step,length);

		length=step;

		step=(step%2==0 || step==1)?(step/2):(step/2 +1);
	}

	CUDA_SAFE_CALL(hipMemcpy(&sum, dDiffData, sizeof(float), hipMemcpyDeviceToHost));

	if( sum < (maxValue - minValue) * size / 10)
		listOrder=NEARLY_SORTED;
	else 
		listOrder=DISORDERLY;

	CUDA_SAFE_CALL(hipFree(dDiffData));
}


void bbSort(float4* dData,int size,int listOrder)
{
	unsigned int timer;

	CUT_SAFE_CALL(cutCreateTimer(&timer));

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	//CUT_SAFE_CALL(cutStartTimer(timer));

	float minValue,maxValue;

	float4*  dTmpData;
	
	
	CUDA_SAFE_CALL(hipMalloc((void**)&dTmpData, sizeof(float4) * size));
	
	CUDA_SAFE_CALL(hipMemcpy(dTmpData, dData, sizeof(float4) * size, hipMemcpyDeviceToDevice));


	reduceMinMax(dTmpData,size,maxValue,true);

	
	

	CUDA_SAFE_CALL(hipMemcpy(dTmpData, dData, sizeof(float4) * size, hipMemcpyDeviceToDevice));

	

	reduceMinMax(dTmpData,size,minValue,false);

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//CUT_SAFE_CALL(cutStopTimer(timer));  

	//float t = cutGetAverageTimerValue(timer);

	//printf( "Max min used: %fms\n",t);

	


	if(minValue == maxValue)
	{
		CUDA_SAFE_CALL(hipFree(dTmpData));
		//printf(" flat, size is:%d\n",size);
		return ;
	}

	if(listOrder == AUTO_EVALUATE )
	{
		evaluateDisorder(dData,size,maxValue,minValue,listOrder);
	}

	

	int blockCount;

	if(size%BLOCK_SIZE==0)blockCount=size/BLOCK_SIZE;
	else blockCount=size/BLOCK_SIZE+1;

	float sliceStep =(float)(50.0*((double)(maxValue-minValue)/(double)size));

	int sliceSize = (int)((maxValue-minValue)/sliceStep) + 10;

	unsigned int* dSliceCounts;
	unsigned int* dOffsetInSlice;

	CUDA_SAFE_CALL(hipMalloc((void**)&dOffsetInSlice, sizeof(unsigned int) * size));
	CUDA_SAFE_CALL(hipMalloc((void**)&dSliceCounts, sizeof(unsigned int) * sliceSize));
	CUDA_SAFE_CALL(hipMemset(dSliceCounts,0, sizeof(int) * sliceSize));

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	//CUT_SAFE_CALL(cutResetTimer(timer));

	if(listOrder == NEARLY_SORTED)
	{
		assignElementToSlicesNearlySortedD<<<blockCount, BLOCK_SIZE>>>(dData,size,dSliceCounts,dOffsetInSlice,minValue,sliceStep,sliceSize,blockCount);
		//printf("NEARLY SORTED\n");
	}
	else 
		assignElementToSlicesD<<<blockCount, BLOCK_SIZE>>>(dData,size,dSliceCounts,dOffsetInSlice,minValue,sliceStep,sliceSize);

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//CUT_SAFE_CALL(cutStopTimer(timer));  

	//t = cutGetAverageTimerValue(timer);

	//printf( "slice dvision used: %fms\n",t);

	unsigned int* hSliceCounts=new unsigned int[sliceSize];
	CUDA_SAFE_CALL(hipMemcpy(hSliceCounts, dSliceCounts, sizeof(unsigned int) * sliceSize, hipMemcpyDeviceToHost));

	int looseBucketSize=size/100;

	unsigned int* hBucketOffsets=new unsigned int[looseBucketSize];
	unsigned int* hBucketSizes=new unsigned int[looseBucketSize];
	unsigned int* hBucketOfSlices=new unsigned int[sliceSize];
	unsigned int* hSliceOffsetInBucket=new unsigned int[sliceSize];
	int bucketsCount=0;

	memset(hBucketSizes,0,sizeof(int) * looseBucketSize);
	memset(hSliceOffsetInBucket,0,sizeof(unsigned int) * sliceSize);

	bool overflow;

	

	overflow = assignSliceToBuckets(hSliceCounts,sliceSize,hBucketOffsets,hBucketOfSlices,hBucketSizes,hSliceOffsetInBucket,bucketsCount,sliceStep);

	
	
	unsigned int* dBucketOffsets;
	unsigned int* dBucketSizes;

	unsigned int* dBucketOfSlices;
	unsigned int* dSliceOffsetInBucket;

	CUDA_SAFE_CALL(hipMalloc((void**)&dBucketOfSlices, sizeof(unsigned int) * sliceSize));
	CUDA_SAFE_CALL(hipMalloc((void**)&dSliceOffsetInBucket, sizeof(unsigned int) * sliceSize));
	CUDA_SAFE_CALL(hipMalloc((void**)&dBucketOffsets, sizeof(unsigned int) * bucketsCount));
	CUDA_SAFE_CALL(hipMalloc((void**)&dBucketSizes, sizeof(unsigned int) * bucketsCount));


	CUDA_SAFE_CALL(hipMemcpy(dBucketOfSlices, hBucketOfSlices, sizeof(unsigned int) * sliceSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dSliceOffsetInBucket, hSliceOffsetInBucket, sizeof(unsigned int) * sliceSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dBucketOffsets, hBucketOffsets, sizeof(unsigned int) * bucketsCount, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dBucketSizes, hBucketSizes, sizeof(unsigned int) * bucketsCount, hipMemcpyHostToDevice));

	hipBindTexture(0,tBucketOffsets,dBucketOffsets);
	hipBindTexture(0,tBucketSizes,dBucketSizes);
	hipBindTexture(0,tBucketOfSlices,dBucketOfSlices);
	hipBindTexture(0,tSliceOffsetInBucket,dSliceOffsetInBucket);

	

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	//CUT_SAFE_CALL(cutResetTimer(timer));

	assignElementToBucketD<<<blockCount, BLOCK_SIZE>>>(dData,dTmpData,size,dOffsetInSlice,minValue,sliceStep);

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//CUT_SAFE_CALL(cutStopTimer(timer));  

	//t = cutGetAverageTimerValue(timer);

	//printf( "bucket dvision used: %fms\n",t);

	


	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	bitonicSortD<<<bucketsCount, BLOCK_SIZE, sizeof(float4) * BLOCK_SIZE>>>(dTmpData);

    CUDA_SAFE_CALL(hipMemcpy(dData, dTmpData, sizeof(float4) * size, hipMemcpyDeviceToDevice));

	
	
	if(overflow){
		for(int i=0;i<bucketsCount;i++)
		{
			if(hBucketSizes[i] > BLOCK_SIZE)
			{
				bbSort(dData + hBucketOffsets[i],hBucketSizes[i],listOrder);
			}
		}
	}

	
	
	delete hBucketOffsets;
	delete hBucketOfSlices;
	delete hSliceCounts;
	delete hBucketSizes;

	CUDA_SAFE_CALL(hipFree(dOffsetInSlice));
	CUDA_SAFE_CALL(hipFree(dSliceCounts));
	CUDA_SAFE_CALL(hipFree(dTmpData));

	hipUnbindTexture( tBucketSizes );
	CUDA_SAFE_CALL(hipFree(dBucketSizes));

	hipUnbindTexture( tBucketOffsets );
	CUDA_SAFE_CALL(hipFree(dBucketOffsets));

	hipUnbindTexture( tBucketOfSlices );
	CUDA_SAFE_CALL(hipFree(dBucketOfSlices));

	hipUnbindTexture( tSliceOffsetInBucket );
	CUDA_SAFE_CALL(hipFree(dSliceOffsetInBucket));
}
